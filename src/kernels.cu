#include "hip/hip_runtime.h"
#include <vector>

#include "../tester/utils.h"
#include <stdexcept>
#include <algorithm>
#include <vector>
#include <cmath>
#include <limits>
#include <cassert>

#include "flash_attention.cuh"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
/**
 * @brief Find the k-th largest element in a vector using CUDA.
 * 
 * @tparam T Type of elements in the input vector (should support `int` and `float`).
 * @param h_input Host-side input vector.
 * @param k 1-based index of the element to find (e.g., `k=1` returns the largest element).
 * @return T The k-th largest element in `h_input`.

 * @note Must use CUDA kernels for all compute-intensive steps; no significant CPU allowed.
 * @note Library functions that can directly complete a significant part of the work are NOT allowed. 
 * @note For invalid cases, return T(-100).
 * @note Handles device memory management (allocate/copy/free) internally. Errors should be thrown.
 */
template <typename T>
__device__ void partition3(T* data, int low, int high, int& left_eq, int& right_eq) {
    T pivot = data[high];
    int i = low;
    int lt = low;       // data[low..lt-1] > pivot
    int gt = high;      // data[gt+1..high] < pivot
    while (i <= gt) {
        if (data[i] > pivot) { // 大于pivot
            T tmp = data[lt];
            data[lt] = data[i];
            data[i] = tmp;
            lt++; i++;
        } else if (data[i] < pivot) { // 小于pivot
            T tmp = data[i];
            data[i] = data[gt];
            data[gt] = tmp;
            gt--;
        } else { // 等于pivot
            i++;
        }
    }
    left_eq = lt;
    right_eq = gt;
}


template <typename T>
__device__ int quickSelect3(T* data, int low, int high, int k) {
    while (low <= high) {
        int left_eq, right_eq;
        partition3(data, low, high, left_eq, right_eq);
        if (k >= left_eq && k <= right_eq) {
            return k; // 找到第k大元素，k在等于pivot的范围内
        } else if (k < left_eq) {
            high = left_eq - 1; // 在左边找更大的元素
        } else {
            low = right_eq + 1; // 在右边找更小的元素
        }
    }
    return -1; // 没找到，理论上不该出现
}


template <typename T>
__global__ void quickSelectKernel(T* data, int low, int high, int k, int* result_idx) {
    if (threadIdx.x == 0 && blockIdx.x == 0) { // 只用一个线程执行
        int idx = quickSelect3(data, low, high, k);
        *result_idx = idx;
    }
}

template <typename T>
T kthLargest(const std::vector<T>& h_input, size_t k) {
    int n = h_input.size();
    if (k <= 0 || k > n) {
        return T(-100);
    }

    T* d_data;
    hipMalloc(&d_data, n * sizeof(T));
    hipMemcpy(d_data, h_input.data(), n * sizeof(T), hipMemcpyHostToDevice);

    int* d_result_idx;
    int h_result_idx = -1;
    hipMalloc(&d_result_idx, sizeof(int));
    hipMemcpy(d_result_idx, &h_result_idx, sizeof(int), hipMemcpyHostToDevice);

    int kIndex = k - 1;

    quickSelectKernel<T><<<1, 1>>>(d_data, 0, n - 1, kIndex, d_result_idx);
    hipDeviceSynchronize();

    hipMemcpy(&h_result_idx, d_result_idx, sizeof(int), hipMemcpyDeviceToHost);

    T result;
    if (h_result_idx >= 0) {
        hipMemcpy(&result, d_data + h_result_idx, sizeof(T), hipMemcpyDeviceToHost);
    } else {
        result = T(-100);
    }

    hipFree(d_data);
    hipFree(d_result_idx);
    return result;
}
// 用partition和quickSelect函数实现并行化的快速选择算法
// 将数据从主机内存复制到设备内存，然后在GPU上计算第k大的元素
// 最后将结果从设备内存复制回主机内存，并释放设备内存
// 使用CUDA编程需要配置一个NVIDIA GPU环境和CUDA编译器
// CUDA编程通常针对GPU计算密集型任务。

/**
 * @brief Computes flash attention for given query, key, and value tensors.
 * 
 * @tparam T Data type (float) for input/output tensors
 * @param[in] h_q Query tensor of shape [batch_size, tgt_seq_len, query_heads, head_dim]
 * @param[in] h_k Key tensor of shape [batch_size, src_seq_len, kv_heads, head_dim]
 * @param[in] h_v Value tensor of shape [batch_size, src_seq_len, kv_heads, head_dim]
 * @param[out] h_o Output attention tensor of shape [batch_size, tgt_seq_len, query_heads, head_dim]
 * @param[in] batch_size Batch dimension size
 * @param[in] target_seq_len Target sequence length
 * @param[in] src_seq_len Source sequence length  
 * @param[in] query_heads Number of query attention heads
 * @param[in] kv_heads Number of key/value heads (supports grouped query attention)
 * @param[in] head_dim Dimension size of each attention head
 * @param[in] is_causal Whether to apply causal masking
 */
// ---------------------
// Flash Attention Kernel
// ---------------------

// 简单GPU端softmax函数
__device__ void softmax_device(float* scores, int len) {
    float max_val = scores[0];
    for (int i = 1; i < len; ++i) {
        max_val = fmaxf(max_val, scores[i]);
    }

    float sum = 0.0f;
    for (int i = 0; i < len; ++i) {
        scores[i] = expf(scores[i] - max_val);
        sum += scores[i];
    }
    float inv_sum = 1.0f / (sum + 1e-8f);
    for (int i = 0; i < len; ++i) {
        scores[i] *= inv_sum;
    }
}

// 线程块：一个batch一个query head负责一个block，线程负责head_dim
// 由于head_dim一般较大，这里使用threadIdx.x控制head_dim维度循环
// blockIdx.x 控制 batch, blockIdx.y 控制 query_heads
__global__ void flashAttentionKernel(const float* __restrict__ q,
                                     const float* __restrict__ k,
                                     const float* __restrict__ v,
                                     float* __restrict__ o,
                                     int batch_size, int tgt_len, int src_len,
                                     int query_heads, int kv_heads, int head_dim,
                                     bool is_causal) {
    int b = blockIdx.x;    // batch index
    int h = blockIdx.y;    // query head index

    if (b >= batch_size || h >= query_heads) return;

    int kvh = h * kv_heads / query_heads;
    if (kvh >= kv_heads) return;

    extern __shared__ float shared_mem[];
    float* scores = shared_mem;       // size src_len
    // probs可直接用scores覆盖，节省共享内存，这里不额外申请

    for (int tq = 0; tq < tgt_len; ++tq) {
        // 计算score
        for (int sk = threadIdx.x; sk < src_len; sk += blockDim.x) {
            if (is_causal && sk > tq) {
                scores[sk] = -1e9f;
            } else {
                float dot = 0.f;
                for (int d = 0; d < head_dim; ++d) {
                    size_t q_idx = ((size_t)b * tgt_len + tq) * query_heads * head_dim + h * head_dim + d;
                    size_t k_idx = ((size_t)b * src_len + sk) * kv_heads * head_dim + kvh * head_dim + d;
                    dot += q[q_idx] * k[k_idx];
                }
                scores[sk] = dot / sqrtf((float)head_dim);
            }
        }

        __syncthreads();

        // 共享内存中的scores已完全写入，使用单线程计算softmax
        if (threadIdx.x == 0) {
            softmax_device(scores, src_len);
        }

        __syncthreads();

        // 计算加权v
        for (int d = threadIdx.x; d < head_dim; d += blockDim.x) {
            float acc = 0.f;
            for (int sk = 0; sk < src_len; ++sk) {
                size_t v_idx = ((size_t)b * src_len + sk) * kv_heads * head_dim + kvh * head_dim + d;
                acc += scores[sk] * v[v_idx];
            }
            size_t o_idx = ((size_t)b * tgt_len + tq) * query_heads * head_dim + h * head_dim + d;
            o[o_idx] = acc;
        }
        __syncthreads();
    }
}

// ---------------------
// Host template function flashAttention
// ---------------------

template <typename T>
void flashAttention(const std::vector<T>& h_q, const std::vector<T>& h_k,
                    const std::vector<T>& h_v, std::vector<T>& h_o,
                    int batch_size, int target_seq_len, int src_seq_len,
                    int query_heads, int kv_heads, int head_dim, bool is_causal) {
    static_assert(std::is_same<T, float>::value, "Only float supported");

    assert(h_q.size() == static_cast<size_t>(batch_size * target_seq_len * query_heads * head_dim));
    assert(h_k.size() == static_cast<size_t>(batch_size * src_seq_len * kv_heads * head_dim));
    assert(h_v.size() == static_cast<size_t>(batch_size * src_seq_len * kv_heads * head_dim));

    h_o.resize(batch_size * target_seq_len * query_heads * head_dim);

    float *d_q, *d_k, *d_v, *d_o;
    size_t q_size = h_q.size() * sizeof(float);
    size_t k_size = h_k.size() * sizeof(float);
    size_t v_size = h_v.size() * sizeof(float);
    size_t o_size = h_o.size() * sizeof(float);

    hipMalloc(&d_q, q_size);
    hipMalloc(&d_k, k_size);
    hipMalloc(&d_v, v_size);
    hipMalloc(&d_o, o_size);

    hipMemcpy(d_q, h_q.data(), q_size, hipMemcpyHostToDevice);
    hipMemcpy(d_k, h_k.data(), k_size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v.data(), v_size, hipMemcpyHostToDevice);

    dim3 grid(batch_size, query_heads);
    int block_dim = 256;  // 线程数，调优用
    size_t shared_mem_size = src_seq_len * sizeof(float);  // 只用一段共享内存存储scores

    flashAttentionKernel<<<grid, block_dim, shared_mem_size>>>(
        d_q, d_k, d_v, d_o,
        batch_size, target_seq_len, src_seq_len,
        query_heads, kv_heads, head_dim,
        is_causal
    );

    hipDeviceSynchronize();

    hipMemcpy(h_o.data(), d_o, o_size, hipMemcpyDeviceToHost);

    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_o);
}
// *********************************************************************
// Explicit Template Instantiations (REQUIRED FOR LINKING WITH TESTER.O)
// DO NOT MODIFY THIS SECTION
// *********************************************************************
template int kthLargest<int>(const std::vector<int>&, size_t);
template float kthLargest<float>(const std::vector<float>&, size_t);
template void flashAttention<float>(const std::vector<float>&, const std::vector<float>&,
  const std::vector<float>&, std::vector<float>&,
  int, int, int, int, int, int, bool);